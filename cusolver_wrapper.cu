#include <iostream>
#include <eigen3/Eigen/Dense>
#include <eigen3/Eigen/Sparse>

// CUDA stuff
#include "hipsolver.h"
#include "hipsolver.h"
#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"
#include "helper_cusolver.h"

#include "boost/timer/timer.hpp"

class CUSolverWrapper {
public:

private:

};
